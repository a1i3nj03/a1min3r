#include "hip/hip_runtime.h"
/**
 * X16R algorithm (X16 with Randomized chain order)
 *
 * tpruvot 2018 - GPL code
 * a1i3nj03 2018
 *** Uses many of Alexis78's very good kernels ***
 */
/*
compute_70, sm_70
compute_62, sm_62
compute_61, sm_61 //
compute_60, sm_60
compute_52, sm_52 //
compute_50, sm_50
*/
#include <stdio.h>
#include <memory.h>
#include <unistd.h>

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_sha2.h"

}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x16r.h"

hipStream_t streamx[MAX_GPUS];
hipStream_t streamk[MAX_GPUS];

#define GPU_HASH_CHECK_LOG 0
static uint32_t *d_hash[MAX_GPUS];
static int *d_ark[MAX_GPUS];

enum Algo {
	BLAKE = 0,
	BMW,
	GROESTL,
	JH,
	KECCAK,
	SKEIN,
	LUFFA,
	CUBEHASH,
	SHAVITE,
	SIMD,
	ECHO,
	HAMSI,
	FUGUE,
	SHABAL,
	WHIRLPOOL,
	SHA512,
	HASH_FUNC_COUNT
};

static const char* algo_strings[] = {
	"blake",
	"bmw512",
	"groestl",
	"jh512",
	"keccak",
	"skein",
	"luffa",
	"cube",
	"shavite",
	"simd",
	"echo",
	"hamsi",
	"fugue",
	"shabal",
	"whirlpool",
	"sha512",
	NULL
};

static __thread uint32_t s_ntime = UINT32_MAX;
static __thread bool s_implemented = false;
static __thread char hashOrder[HASH_FUNC_COUNT + 1] = { 0 };

//__host__ extern void ark_init(int thr_id);
__host__ void ark_switch(int thr_id);
__host__ int ark_reset(int thr_id);
//__constant__ int arks[MAX_GPUS];
//__constant__ int *d_ark[MAX_GPUS] = { NULL };
//__device__ __constant__ int d_ark[MAX_GPUS];

static void(*pAlgo64[16])(int, uint32_t, uint32_t*, volatile int*) =
{
	quark_blake512_cpu_hash_64,		//2,//TOP_SPEED,	//18.0 > 14 //60
	quark_bmw512_cpu_hash_64,		//1,//TOP_SPEED,	//21.5 > 15 //71
	quark_groestl512_cpu_hash_64,	//3,//MIN_SPEED,	//2.4  > 14 //7.8
	quark_jh512_cpu_hash_64,		//3,//MID_SPEED,	//8.1  > 13 //24.7
	quark_keccak512_cpu_hash_64,	//1,//TOP_SPEED,	//24.3 > 18 //66.00
	quark_skein512_cpu_hash_64,		//0,//TOP_SPEED,	//27.1 > 18 //71.5
	x11_luffa512_cpu_hash_64_alexis,//2,//MID_SPEED,	//13   > 18 //32.1
	x11_cubehash512_cpu_hash_64,	//3,//LOW_SPEED,	//7.4  > 18 //17
	x11_shavite512_cpu_hash_64_alexis,//3,LOW_SPEED,	//8    > 18 //14.82
	x11_simd512_cpu_hash_64,		//3,//MIN_SPEED,	//3.5  > 18 //6.08
	x11_echo512_cpu_hash_64_alexis,	//3,//LOW_SPEED,	//4    > 18 //8.7
	x13_hamsi512_cpu_hash_64_alexis,//3,//LOW_SPEED,	//5.1  > 18 //10.6
	x13_fugue512_cpu_hash_64_alexis,//3,//LOW_SPEED,	//6.7  > 19 //11.6
	x14_shabal512_cpu_hash_64_alexis,//0,/TOP_SPEED,	//39   > 18 //115
	x15_whirlpool_cpu_hash_64,		//3,//LOW_SPEED,	//7.0  > 21 //15.8
	x17_sha512_cpu_hash_64			//0//TOP_SPEED	//28.5 > 18 //71
};

static void(*pAlgo80[16])(int, uint32_t, uint32_t, uint32_t*, volatile int*) =
{
	quark_blake512_cpu_hash_80,
	quark_bmw512_cpu_hash_80,
	groestl512_cuda_hash_80,
	jh512_cuda_hash_80,
	keccak512_cuda_hash_80,
	skein512_cpu_hash_80,
	qubit_luffa512_cpu_hash_80_alexis,
	cubehash512_cuda_hash_80,
	x11_shavite512_cpu_hash_80,
	x16_simd512_cuda_hash_80,
	x16_echo512_cuda_hash_80,
	x16_hamsi512_cuda_hash_80,
	x16_fugue512_cuda_hash_80,
	x16_shabal512_cuda_hash_80,
	x16_whirlpool512_hash_80,
	x16_sha512_cuda_hash_80
};

/*
BLAKE = 0,
BMW,1
GROESTL,2
JH,3
KECCAK,4
SKEIN,5
LUFFA,6
CUBEHASH,7
SHAVITE,8
SIMD,9
ECHO,a
HAMSI,b
FUGUE,c
SHABAL,d
WHIRLPOOL,e
SHA512,f
*/

static void getAlgoString(const uint32_t* prevblock, char *output)
{
	for (int i = 0; i < 16; i++)
	{
			*output++ = (*(uint64_t*)prevblock >> 60 - (i * 4)) & 0x0f;
	}
}

// X16R CPU Hash (Validation)
extern "C" void x16r_hash(void *output, const void *input)
{
	//unsigned char _ALIGN(64) hash[128];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;

	void *in = (void*) input;
	int size = 80;

	uint32_t *in32 = (uint32_t*) input;
	uint64_t prevblock = *(uint64_t*)&in32[1];

	for (int i = 0; i < 16; i++)
	{

		switch ((prevblock >> 60 - (i << 2)) & 0x0f) {
		case BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, in, size);
			sph_blake512_close(&ctx_blake, output);
			break;
		case BMW:
			sph_bmw512_init(&ctx_bmw);
			sph_bmw512(&ctx_bmw, in, size);
			sph_bmw512_close(&ctx_bmw, output);
			break;
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, in, size);
			sph_groestl512_close(&ctx_groestl, output);
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, in, size);
			sph_skein512_close(&ctx_skein, output);
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, in, size);
			sph_jh512_close(&ctx_jh, output);
			break;
		case KECCAK:
			sph_keccak512_init(&ctx_keccak);
			sph_keccak512(&ctx_keccak, in, size);
			sph_keccak512_close(&ctx_keccak, output);
			break;
		case LUFFA:
			sph_luffa512_init(&ctx_luffa);
			sph_luffa512(&ctx_luffa, in, size);
			sph_luffa512_close(&ctx_luffa, output);
			break;
		case CUBEHASH:
			sph_cubehash512_init(&ctx_cubehash);
			sph_cubehash512(&ctx_cubehash, in, size);
			sph_cubehash512_close(&ctx_cubehash, output);
			break;
		case SHAVITE:
			sph_shavite512_init(&ctx_shavite);
			sph_shavite512(&ctx_shavite, in, size);
			sph_shavite512_close(&ctx_shavite, output);
			break;
		case SIMD:
			sph_simd512_init(&ctx_simd);
			sph_simd512(&ctx_simd, in, size);
			sph_simd512_close(&ctx_simd, output);
			break;
		case ECHO:
			sph_echo512_init(&ctx_echo);
			sph_echo512(&ctx_echo, in, size);
			sph_echo512_close(&ctx_echo, output);
			break;
		case HAMSI:
			sph_hamsi512_init(&ctx_hamsi);
			sph_hamsi512(&ctx_hamsi, in, size);
			sph_hamsi512_close(&ctx_hamsi, output);
			break;
		case FUGUE:
			sph_fugue512_init(&ctx_fugue);
			sph_fugue512(&ctx_fugue, in, size);
			sph_fugue512_close(&ctx_fugue, output);
			break;
		case SHABAL:
			sph_shabal512_init(&ctx_shabal);
			sph_shabal512(&ctx_shabal, in, size);
			sph_shabal512_close(&ctx_shabal, output);
			break;
		case WHIRLPOOL:
			sph_whirlpool_init(&ctx_whirlpool);
			sph_whirlpool(&ctx_whirlpool, in, size);
			sph_whirlpool_close(&ctx_whirlpool, output);
			break;
		case SHA512:
			sph_sha512_init(&ctx_sha512);
			sph_sha512(&ctx_sha512,(const void*) in, size);
			sph_sha512_close(&ctx_sha512, (void*)output);
			break;
		}
		in = (void*) output;
		size = 64;
	}
//	memcpy(output, hash, 32);
}

void whirlpool_midstate(void *state, const void *input)
{
	sph_whirlpool_context ctx;

	sph_whirlpool_init(&ctx);
	sph_whirlpool(&ctx, input, 64);

	memcpy(state, ctx.state, 64);
}

static bool init[MAX_GPUS] = { 0 };

extern volatile int init_items[MAX_GPUS];
volatile int *volatile h_ark[MAX_GPUS] = { NULL };
extern pthread_mutex_t ark_lock;

//#define _DEBUG
#define _DEBUG_PREFIX "x16r-"
#include "cuda_debug.cuh"

#if GPU_HASH_CHECK_LOG == 1
static int algo80_tests[HASH_FUNC_COUNT] = { 0 };
static int algo64_tests[HASH_FUNC_COUNT] = { 0 };
#endif
static int algo80_fails[HASH_FUNC_COUNT] = { 0 };
#define NO_ORDER_COUNTER 1
#define BOOST 0//0x10000

__global__ void set_hi(int *ark)
{
	*ark = 1;
}

__global__ void set_lo(int *ark)
{
	*ark = 0;
}

#define TOP_SPEED 0
#define MID_SPEED 1
#define LOW_SPEED 3
#define MIN_SPEED 6
#define SIMD_MAX (3 << 19)
uint8_t target_table[16] =
{
	//18 ,21 ,2.5,8 ,24 ,27 ,13,7.5,8 ,3.5,4 ,5 ,6.5,39,7 ,28.5
	//.45,.55,.1 ,.2,.55,.70,.3,.2 ,.2,.1 ,.1,.1,.2,1  ,.2,.7
	//4,5,1,2,5,7,3,2,2,1,1,1,2,10,2,7
	//6,5,9,8,5,3,7,8,8,9,9,9,9, 0,8,3
	//3,2,4,4,2,1,3,4,4,4,4,4,4, 0,4,1

	6,//TOP_SPEED,	//18.0 > 14 //60
	5,//TOP_SPEED,	//21.5 > 15 //71
	9,//MIN_SPEED,	//2.4  > 14 //7.8
	8,//MID_SPEED,	//8.1  > 13 //24.7
	5,//TOP_SPEED,	//24.3 > 18 //66.00
	3,//TOP_SPEED,	//27.1 > 18 //71.5
	7,//MID_SPEED,	//13   > 18 //32.1
	8,//LOW_SPEED,	//7.4  > 18 //17
	8,//LOW_SPEED,	//8    > 18 //14.82
	9,//MIN_SPEED,	//3.5  > 18 //6.08
	9,//LOW_SPEED,	//4    > 18 //8.7
	9,//LOW_SPEED,	//5.1  > 18 //10.6
	9,//LOW_SPEED,	//6.7  > 19 //11.6
	0,//TOP_SPEED,	//39   > 18 //115
	8,//LOW_SPEED,	//7.0  > 21 //15.8
	3//TOP_SPEED	//28.5 > 18 //71
};

static uint32_t max_throughput = 0;

void target_throughput(uint64_t target, uint32_t &throughput)
{
	bool simd = 0;
	uint32_t t = throughput;
	int avg = target_table[(target >> 60) & 0x0f];
	if (((target >> 60) & 0x0f) == SIMD)
		simd = 1;
	for (int i = 1; i < 16; i++)
	{
		avg += target_table[(target >> 60 - (i << 2)) & 0x0f];
		if (((target >> 60 - (i << 2)) & 0x0f) == SIMD)
			simd = 1;
	}
//	applog(LOG_DEBUG, "%d >> 4 = %d", avg, avg >> 4);
	int ratio;
	if (throughput >= 1 << 31)
		ratio = 10;
	if (throughput >= 1 << 30)
		ratio = 11;
	if (throughput >= 1 << 29)
		ratio = 12;
	else if (throughput >= 1 << 28)
		ratio = 13;
	else if (throughput >= 1 << 27)
		ratio = 14;
	else if (throughput >= 1 << 26)
		ratio = 15;
	else if (throughput >= 1 << 25)
		ratio = 16;
	else if (throughput >= 1 << 24)
		ratio = 20;
	else if (throughput >= 1 << 23)
		ratio = 24;
	else if (throughput >= 1 << 22)
		ratio = 28;
	else if (throughput >= 1 << 21)
		ratio = 32;
	else if (throughput >= 1 << 20)
		ratio = 36;
	else if (throughput >= 1 << 19)
		ratio = 40;
	else if (throughput >= 1 << 18)
		ratio = 44;
	else if (throughput >= 1 << 17)
		ratio = 48;
	else
		ratio = avg | 1;

	avg += (-avg % ratio) > 0 ? (-avg % ratio) : -(-avg % ratio);
	throughput >>= (avg / ratio);
	throughput += -(int)throughput & 0xfff;
//	throughput = (t < throughput) ? t : throughput;
	throughput = (simd && (throughput >(SIMD_MAX))) ? SIMD_MAX : throughput;
	throughput = (throughput) ? throughput : 0x1000;
	throughput = (throughput <= max_throughput)? throughput : max_throughput;
}

extern "C" int x16r_init(int thr_id, uint32_t max_nonce)
{
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21) + BOOST;
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync | hipDeviceMapHost);
		}

		max_throughput = throughput;
		if (max_throughput > (1 << 21))
			throughput = 1 << 21;
		while (hipMalloc(&d_hash[thr_id], (size_t)64 * throughput) != hipSuccess)
		{
			throughput >>= 1;
			throughput -= 0x4000;
			throughput &= ~0x3fff;
			if (throughput < (1 << 14))
				CUDA_CALL_OR_RET_X(hipErrorOutOfMemory, 0);
		}
		if (max_throughput != throughput)
			gpulog(LOG_INFO, thr_id, "Intensity adjusted to %g, %u cuda threads", throughput2intensity(throughput - BOOST), throughput - BOOST);
		else
			gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput - BOOST), throughput - BOOST);

		max_throughput = throughput;

		CUDA_SAFE_CALL(hipHostMalloc((void **)&h_ark[thr_id], sizeof(int)*16));
		CUDA_CALL_OR_RET_X(hipMalloc(&d_ark[thr_id], sizeof(int)*16), 0);

//		CUDA_SAFE_CALL(hipMalloc(&d_ark[thr_id], sizeof(int)));
		*h_ark[thr_id] = 0;
//		if (thr_id == 0)
		{
//			CUDA_SAFE_CALL(hipStreamCreate(&streamx[0]));
//			CUDA_SAFE_CALL(hipStreamCreate(&streamk[0]));
//			CUDA_SAFE_CALL(hipStreamCreateWithPriority(&streamk[0], 0, 0));
			CUDA_SAFE_CALL(hipStreamCreateWithPriority(&streamx[thr_id], hipStreamNonBlocking, -1));
		}
//		else
		{
//			while (h_ark[0] == NULL)
//				sleep(1);
		}
//		set_lo << <1, 1 >> >(d_ark[thr_id]);
		CUDA_SAFE_CALL(hipMemcpy(d_ark[thr_id], (int*)h_ark[thr_id], sizeof(int)*16, hipMemcpyHostToDevice));
//		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_ark[thr_id]), (int*)h_ark[thr_id], sizeof(int), 0, hipMemcpyHostToDevice));
		
		//		CUDA_SAFE_CALL(hipGetLastError());
		//		CUDA_SAFE_CALL(hipStreamSynchronize(streamx[thr_id]));

		pthread_mutex_lock(&ark_lock);
		init_items[thr_id] = 1;
		pthread_mutex_unlock(&ark_lock);

		//		ark_init(thr_id);
//		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput - BOOST), throughput - BOOST);
#if 0
		if (throughput2intensity(throughput - BOOST) > 21) gpulog(LOG_INFO, thr_id, "SIMD throws error on malloc call, TBD if there is a fix");
#endif
		/*
		BLAKE = 0,
		BMW,1
		GROESTL,2
		JH,3
		KECCAK,4
		SKEIN,5
		LUFFA,6
		CUBEHASH,7
		SHAVITE,8
		SIMD,9
		ECHO,a
		HAMSI,b
		FUGUE,c
		SHABAL,d
		WHIRLPOOL,e
		SHA512,f
		*/

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
//		x11_shavite512_cpu_init(thr_id, throughput);
		if (throughput > (SIMD_MAX))
		{
			if (x11_simd512_cpu_init(thr_id, SIMD_MAX))
			{
				applog(LOG_WARNING, "SIMD was unable to initialize :( exiting...");
				exit(-1);
			}// 64
		}
		else if (x11_simd512_cpu_init(thr_id, throughput))
		{
			applog(LOG_WARNING, "SIMD was unable to initialize :( exiting...");
			exit(-1);
		}// 64
//		x16_echo512_cuda_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x16_fugue512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x16_whirlpool512_init(thr_id, throughput);
		x17_sha512_cpu_init(thr_id, throughput);

//		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t)64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);
		hipGetLastError();

		init[thr_id] = true;
	}
	return -128;
}
extern volatile time_t g_work_time;

static uint64_t tlast[MAX_GPUS] = { 0 };


extern "C" int scanhash_x16r(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, uint64_t seq)
{
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21) + BOOST;
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	static uint32_t compute_throughput;
	static int retry_target = 0;
	if (pdata[19] == max_nonce)
	{
		if (seq == ~0ULL)
			*hashes_done = pdata[19] - first_nonce + throughput;
		return -128;
	}
	//	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
	//	if (strstr(device_name[dev_id], "GTX 1080")) intensity = 20;
	//	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21);
	int g_work_signal = 0;
	uint32_t _ALIGN(64) endiandata[20];

	if (opt_benchmark) {
		/*
		((uint32_t*)ptarget)[7] = 0x003f;
		((uint32_t*)pdata)[1] = 0x88888888;
		((uint32_t*)pdata)[2] = 0x88888888;
		//! Should cause vanila v0.1 code to have shavite CPU invalid hash error at various intervals
		*/
		((uint32_t*)ptarget)[7] = 0x123f; // 2:64/80 + D:64  broke
		*((uint64_t*)&pdata[1]) = 0x2222222000000000;//seq;//0x67452301EFCDAB89;//0x31C8B76F520AEDF4;
		//		*((uint64_t*)&pdata[1]) = 0xbbbbbbbbbbbbbbbb;//2:64,4:80,8,a,e.. error//44B54B9F248C0708//0x31C8B76F520AEDF4;
		//489f 4f38 33f4 7016 //01346789f
//		((uint32_t*)pdata)[17] = 0x12345678;


	}

	for (int k = 0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	if (tlast[thr_id] != (*(uint64_t*)&endiandata[1]))
	{
		if (!thr_id)
		{

			target_throughput(*(uint64_t*)&endiandata[1], throughput);
			applog(LOG_INFO, "[%08X%08X] (%08X) (%f)", endiandata[2], endiandata[1], swab32(pdata[17]), throughput2intensity(throughput));
			tlast[0] = (*(uint64_t*)&endiandata[1]);
		}
		else
		{
			tlast[thr_id] = (*(uint64_t*)&endiandata[1]);
			target_throughput(*(uint64_t*)&endiandata[1], throughput);
		}
		compute_throughput = throughput;
		throughput = min(throughput, max_nonce - first_nonce);
	}
	else
		throughput = min(compute_throughput, max_nonce - first_nonce);

	/*
	if (throughput >= ((max_nonce - first_nonce) >> 1))
	{
		if (seq == ~0ULL)
			*hashes_done = pdata[19] - first_nonce + throughput;
		return -128; // free hashes
	}
	*/

	uint8_t algo80;

	cuda_check_cpu_setTarget(ptarget, thr_id);

	algo80 = (*(uint64_t*)&endiandata[1] >> 60) & 0x0f;
	switch (algo80) {
	case BLAKE:
		//! low impact, can do a lot to optimize quark_blake512
		quark_blake512_cpu_setBlock_80(thr_id, endiandata);
		break;
	case BMW:
		//! low impact, painfully optimize quark_bmw512
		quark_bmw512_cpu_setBlock_80(thr_id, endiandata);
		break;
	case GROESTL:
		//! second most used algo historically
		groestl512_setBlock_80(thr_id, endiandata);
		break;
	case JH:
		//! average use, optimization tbd
		jh512_setBlock_80(thr_id, endiandata);
		break;
	case KECCAK:
		//! low impact
		keccak512_setBlock_80(thr_id, endiandata);
		break;
	case SKEIN:
		//! very low impact
		skein512_cpu_setBlock_80(thr_id, (void*)endiandata);
		break;
	case LUFFA:
		//! moderate impact (more than shavite)
		qubit_luffa512_cpu_setBlock_80_alexis(thr_id, (void*)endiandata);
		break;
	case CUBEHASH:
		//! moderate impact (more than shavite)
		cubehash512_setBlock_80(thr_id, endiandata);
		break;
	case SHAVITE:
		//! has been optimized fairly well
		x11_shavite512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case SIMD:
		//! high impact optimization. -i > 21 causes error.
		x16_simd512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case ECHO:
		//! high impact needs more optimizations
		x16_echo512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case HAMSI:
		//! ***highest impact***
		x16_hamsi512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case FUGUE:
		//! very high impact!
		x16_fugue512_setBlock_80(thr_id, (void*)pdata);
		break;
	case SHABAL:
		//! very low impact.
		x16_shabal512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case WHIRLPOOL:
		//! moderate impact (more than shavite by a bit)
		x16_whirlpool512_setBlock_80(thr_id, (void*)endiandata);
		break;
	case SHA512:
		//! second lowest impact.
		x16_sha512_setBlock_80(thr_id, endiandata);
		break;
	}

//	work->nonces[0] = UINT32_MAX;
	int warn = 0;

	do {
		pAlgo80[(*(uint64_t*)&endiandata[1] >> 60 - (0 * 4)) & 0x0f](thr_id, throughput, pdata[19], d_hash[thr_id], d_ark[thr_id]);
//		hipStreamSynchronize(streamx[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (1 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (2 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (3 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (4 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (5 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (6 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (7 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (8 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (9 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (10 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (11 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (12 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (13 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (14 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);
		pAlgo64[(*(uint64_t*)&endiandata[1] >> 60 - (15 * 4)) & 0x0f](thr_id, throughput, d_hash[thr_id], d_ark[thr_id]);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id], d_ark[thr_id]);
#ifdef _DEBUG
		uint32_t _ALIGN(64) dhash[8];
		be32enc(&endiandata[19], pdata[19]);
		x16r_hash(dhash, endiandata);
		applog_hash(dhash);
		return -1;
#endif
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			x16r_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1, d_ark[thr_id]);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					x16r_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				}
				else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#if GPU_HASH_CHECK_LOG == 1
				gpulog(LOG_INFO, thr_id, "hash found with %s 80!", algo_strings[algo80]);

				algo80_tests[algo80] += work->valid_nonces;
				char oks64[128] = { 0 };
				char oks80[128] = { 0 };
				char fails[128] = { 0 };
				for (int a = 0; a < HASH_FUNC_COUNT; a++) {
//					const char elem = hashOrder[a];
					const uint8_t algo64 = (*(uint64_t*)&endiandata[1] >> 60 - (a * 4)) & 0x0f;//elem >= 'A' ? elem - 'A' + 10 : elem - '0';
					if (a > 0) algo64_tests[algo64] += work->valid_nonces;
					sprintf(&oks64[strlen(oks64)], "|%X:%2d", a, algo64_tests[a] < 100 ? algo64_tests[a] : 99);
					sprintf(&oks80[strlen(oks80)], "|%X:%2d", a, algo80_tests[a] < 100 ? algo80_tests[a] : 99);
					sprintf(&fails[strlen(fails)], "|%X:%2d", a, algo80_fails[a] < 100 ? algo80_fails[a] : 99);
				}
				applog(LOG_INFO, "K64: %s", oks64);
				applog(LOG_INFO, "K80: %s", oks80);
				applog(LOG_ERR,  "F80: %s", fails);
#endif
				if (ark_reset(thr_id))
				{
//					*hashes_done = 0;//pdata[19] - first_nonce - throughput;
					return -127;
//					return work->valid_nonces;
				}
				//				if (work_restart[thr_id].restart) return -127;
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				// x11+ coins could do some random error, but not on retry
				if (ark_reset(thr_id))
				{
//					*hashes_done = 0;//pdata[19] - first_nonce - throughput;
					return -127;
				}
				gpu_increment_reject(thr_id);
				algo80_fails[algo80]++;
				if (!opt_quiet)	gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! %s %X%X",
					work->nonces[0], algo_strings[algo80], endiandata[2], endiandata[1]);
				if (!warn) {
					warn++;
					pdata[19] = work->nonces[0] + 1;
					continue;
				}
				else {
//					if (!opt_quiet)	gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! %s %X%X",
//						work->nonces[0], algo_strings[algo80], endiandata[2], endiandata[1]);
					//					work->nonces[0], algo_strings[algo80], hashOrder);
					warn = 0;
					//					work->data[19] = max_nonce;
					//					if (work_restart[thr_id].restart) return -127;
					//					return -128;
				}
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			if (pdata[19] == max_nonce)
				break;
//			gpulog(LOG_INFO, thr_id, "G_WORK2");
			/*
			if ((throughput >> 1) > max_nonce - pdata[19])
			{
//				pdata[19] = max_nonce;
				if (ark_reset(thr_id))
				{
					return -127;
				}
				return 0;
			}
			*/
			throughput = max_nonce - pdata[19];
			pdata[19] = max_nonce;
			if (ark_reset(thr_id))
			{
				return -127;
			}
			if (throughput < 0x1000)
				return -127;
				//	if (work_restart[thr_id].restart) return -127;
			continue;
		}
		else
		{

			if (!g_work_signal && throughput >= ((max_nonce - pdata[19]) >> 2))
			{
				g_work_time = 0;
//				gpulog(LOG_INFO, thr_id, "G_WORK3");
			}
			pdata[19] += throughput;
		}
		/*
		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;
		*/
	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart && *h_ark[thr_id] == 0);

	if ((uint64_t)throughput + pdata[19] < max_nonce)
		*hashes_done = pdata[19] - first_nonce;

	if (ark_reset(thr_id))
	{
		return -127;
	}
	//	if (work_restart[thr_id].restart) return -127;
	return 0;
}

// cleanup
extern "C" void free_x16r(int thr_id)
{
	if (!init[thr_id])
		return;
//	ark_switch(thr_id);
	hipDeviceSynchronize();
//	ark_reset(thr_id);
	hipFree(d_hash[thr_id]);
//	hipStreamDestroy(streamk[0]);
	hipStreamDestroy(streamx[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x13_fugue512_cpu_free(thr_id);
	x16_fugue512_cpu_free(thr_id); // to merge with x13_fugue512 ?
	x15_whirlpool_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}


#if 0
__host__
void ark_init(int thr_id)
{
	pthread_mutex_lock(&ark_lock);
	if (q)
	{
		q = 0;
		CUDA_SAFE_CALL(hipHostMalloc((void **)&h_ark, sizeof(int) * MAX_GPUS));
		memset(h_ark, 0, sizeof(int) * MAX_GPUS);
		for (int i = 0; i < MAX_GPUS; i++)
		{
//			CUDA_SAFE_CALL(hipHostAlloc((void **)&h_ark[thr_id], sizeof(int), hipHostMallocPortable));
//			CUDA_SAFE_CALL(hipHostAlloc((void **)&h_ark[thr_id], sizeof(int), hipHostMallocPortable));
//			h_ark[thr_id] = 0;
			CUDA_SAFE_CALL(hipStreamCreate(&streamx[thr_id]));
			CUDA_SAFE_CALL(hipStreamCreate(&streamk[thr_id]));
			CUDA_SAFE_CALL(hipMalloc(&d_ark[thr_id], sizeof(int) * 16));
//			hipMemcpyToSymbol(HIP_SYMBOL(d_ark[thr_id]), (int*)&h_ark[thr_id], sizeof(int), 0, hipMemcpyHostToDevice);
//			CUDA_SAFE_CALL(hipMemcpy(d_ark[thr_id], h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice));
			CUDA_SAFE_CALL(hipMemcpyAsync(d_ark[thr_id], &h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice, streamx[thr_id]));
			CUDA_SAFE_CALL(hipGetLastError());
			//	hipMemcpyAsync(d_ark, (int*)&h_ark, sizeof(int), hipMemcpyHostToDevice, stream1);
		}
		CUDA_SAFE_CALL(hipStreamSynchronize(streamx[thr_id]));
	}
	pthread_mutex_unlock(&ark_lock);
}
//--default-stream per-thread // compute_61,sm_61
#endif
__host__ void ark_switch(int thr_id)
{
//	while (q < thr_id) sleep(1);
	if (init_items[thr_id]) //&& (*h_ark[thr_id] == 0))
	{
		hipSetDevice(device_map[thr_id]);
//		set_hi << <1, 1, 0, streamx[0]>> >(d_ark[thr_id]);
//		CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_ark[thr_id]), (int*)h_ark[thr_id], sizeof(int), 0, hipMemcpyHostToDevice, streamx[0]));
//		if (*h_ark[thr_id] == 0)
		{
			*h_ark[thr_id] = 1;
#ifdef A1MIN3R_MOD
			CUDA_SAFE_CALL(hipMemsetAsync(d_ark[thr_id], 1, 1, streamx[thr_id]));
//			CUDA_SAFE_CALL(hipMemcpyAsync(d_ark[thr_id], (int*)h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice, streamx[0]));
#endif
		}
//		else
		{
#ifdef A1MIN3R_MOD
//			CUDA_SAFE_CALL(hipMemcpyAsync(d_ark[thr_id], (int*)h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice, streamx[0]));
#endif
		}
	}
}
//CUDA_API_PER_THREAD_DEFAULT_STREAM
__host__ int ark_reset(int thr_id)
{
//	hipStreamSynchronize(streamk[thr_id]);
//	pthread_mutex_lock(&ark_lock);
	if (*h_ark[thr_id]) //! Call needs check to avoid 
	{

//		pthread_mutex_unlock(&ark_lock);
		//		CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_ark[thr_id]), (int*)h_ark[thr_id], sizeof(int), 0, hipMemcpyHostToDevice, streamx[thr_id]));
//		CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_ark[thr_id]), (int*)h_ark[thr_id], sizeof(int), 0, hipMemcpyHostToDevice, streamx[thr_id]));
		*h_ark[thr_id] = 0;
#ifdef A1MIN3R_MOD
		CUDA_SAFE_CALL(hipMemsetAsync(d_ark[thr_id], 0, 1, 0));
//		CUDA_SAFE_CALL(hipMemcpyAsync(d_ark[thr_id], (int*)h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice, 0));
#endif
		return 1;
	}
	else
#ifdef A1MIN3R_MOD
//				CUDA_SAFE_CALL(hipMemcpyAsync(d_ark[thr_id], (int*)h_ark[thr_id], sizeof(int), hipMemcpyHostToDevice, 0));
#endif
	//		pthread_mutex_unlock(&ark_lock);
//	CUDA_SAFE_CALL(hipStreamSynchronize(streamx[thr_id]));
	return 0;
}
